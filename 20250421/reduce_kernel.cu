
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_DIM 1024

__global__ void reduce_kernel(float* input, float* output, unsigned int n)
{
	unsigned int segment = blockIdx.x * blockDim.x*2;
	unsigned int i = threadIdx.x*2 + segment;

	for (unsigned int stride = BLOCK_DIM; stride > 0; stride /=2)
	{
		if (threadIdx.x < stride)
		{
			input[i] += input[i + stride];
		}
		__syncthreads();
		if (threadIdx.x == 0)
		{
			output[blockIdx.x] = input[i];
		}
	}
}

void reduce_gpu(float* input, float* output, unsigned int n)
{
	float *input_d;
	float *output_d;
	
    	const unsigned int numThreadsPerBlock = BLOCK_DIM;
    	const unsigned int numElementsPerBlock = numThreadsPerBlock * 2;
    	const unsigned int numBlocks = (n + numElementsPerBlock - 1) / numElementsPerBlock;


	hipMalloc((void**)&input_d, n*sizeof(float));
	hipMalloc((void**)&output_d, numBlocks * sizeof(float));
	hipDeviceSynchronize();

	hipMemcpy(input_d,input,n*sizeof(float),hipMemcpyHostToDevice);
	hipDeviceSynchronize();


	reduce_kernel<<<numBlocks,numThreadsPerBlock>>>(input_d,output_d,n);
	hipDeviceSynchronize();

	hipMemcpy(output,output_d,numBlocks*sizeof(float),hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	hipFree(output_d);
	hipFree(input_d);
}

int main()
{
	unsigned int n = 1000000;

	float *input = (float*)malloc(n * sizeof(float));
	unsigned int numBlocks = (n + BLOCK_DIM*2-1)/(BLOCK_DIM*2);
	float *output = (float*)malloc(numBlocks * sizeof(float));

	for (unsigned int i = 0; i < n; i++) {
		input[i] = 1.0f;
	}

	reduce_gpu(input,output,n);

	float final_sum = 0.0f;
	for (unsigned int i = 0; i < numBlocks; i++){
		final_sum +=output[i];
	}

	free(input);
	free(output);


	
	return 0;
}
